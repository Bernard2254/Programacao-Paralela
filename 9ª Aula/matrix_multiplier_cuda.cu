
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matMult(float *d_A,float *d_B, float *d_C, int n){
        int col = threadIdx.x + (blockDim.x * blockIdx.x);
        int row = threadIdx.y + (blockDim.y*blockIdx.y);
        if(col>=n||row>=n) return;
        float seguraPraMimBernard=0;
        for(int a=0;a<n;a++){
                seguraPraMimBernard+=d_A[row+a*n]*d_B[a+col*n];
        }
        d_C[row+col*n]=seguraPraMimBernard;
}

int main(){
        int N=32;
        int Nthreads = 256;
        int size = N*sizeof(float); float *d_A, *d_B, *d_C;
        float *A, *B, *C;
        A = (float*)malloc(size);
        B = (float*)malloc(size);
        C = (float*)malloc(size);
        int i=0;
        for(i=0; i<N; i++){
                A[i]=i;
                B[i]=i;
        }
        hipMalloc((void **) &d_A, size);
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMalloc((void **) &d_B, size);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
        hipMalloc((void **) &d_C, size);

        dim3 blockSize((N+(Nthreads -1))/Nthreads,(N+(Nthreads -1))/Nthreads,1);
        dim3 thrdPBlock(Nthreads,Nthreads,1);

        matMult<<<blockSize,thrdPBlock>>>(d_A, d_B, d_C, N);

        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost); //É uma função síncrona nativamente
        for(i=0; i<N; i++)
                printf("%f ", C[i]);
        hipFree(d_A);hipFree(d_B);hipFree(d_C);
}