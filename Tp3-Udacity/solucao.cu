#include "hip/hip_runtime.h"
#include <limits.h>

#include <float.h>

#include <math.h>

#include <stdio.h>



#include "utils.h"



#define THREADS 1024



__global__

void histograma(unsigned int* d_bins, const float* d_in, const int bins, const float min, const float max, const int size) {  

    int main_id = threadIdx.x+blockDim.x * blockIdx.x;

    float range = max-min;

    int bin_id = ((d_in[main_id]-min)/range)*bins;

    

    atomicAdd(&d_bins[bin_id], 1);

}



__global__ 

void scan(unsigned int* d_bins, int size) {

    int main_id = threadIdx.x + blockDim.x * blockIdx.x;

    

    for(int s = 1; s <= size; s *= 2) {

          int ind = main_id - s; 

         

          unsigned int val = 0;

          if(ind >= 0)

              val = d_bins[ind];

          __syncthreads();

              d_bins[main_id] += val;

          __syncthreads();

    }

}



__global__

void reduction_kernel(const float* const d_kernel_in, float* d_kernel_out, const size_t size, int minmax) {

    extern __shared__ float shared[];

    

    int main_id = threadIdx.x + blockDim.x * blockIdx.x;

    int thread_id = threadIdx.x; 

    

    if(main_id < size) 

        shared[thread_id] = d_kernel_in[main_id];

    __syncthreads();

       

    for(int s = blockDim.x/2; s > 0; s /= 2) {

        if(thread_id < s) {

            if(minmax == 0)

                shared[thread_id] = min(shared[thread_id], shared[thread_id+s]);

            else

                shared[thread_id] = max(shared[thread_id], shared[thread_id+s]);

        }

        __syncthreads();

    }

    

    if(thread_id == 0)

        d_kernel_out[blockIdx.x] = shared[0];

}



float reduction(const float* const d_in, const size_t size, int minmax) {

    float* d_kernel_in;

    

    hipMalloc(&d_kernel_in, sizeof(float) * size);    

    hipMemcpy(d_kernel_in, d_in, sizeof(float) * size, hipMemcpyDeviceToDevice);





    float* d_kernel_out;



    const int mem_size = sizeof(float)*THREADS;

    dim3 block_reduction = (size+THREADS-1)/THREADS;



    for(size_t kernel_size = THREADS*size - (THREADS-1);  kernel_size >= THREADS; ){

        kernel_size = (kernel_size+THREADS-1)/THREADS;

        hipMalloc(&d_kernel_out, sizeof(float) * (kernel_size+THREADS-1)/THREADS);

        reduction_kernel<<<block_reduction, THREADS, mem_size>>>(d_kernel_in, d_kernel_out, kernel_size, minmax);

        d_kernel_in = d_kernel_out;

    }

    

    float out;

    hipMemcpy(&out, d_kernel_out, sizeof(float), hipMemcpyDeviceToHost);

    return out;

}



void your_histogram_and_prefixsum(const float* const d_logLuminance,

                                  unsigned int* const d_cdf,

                                  float &min_logLum,

                                  float &max_logLum,

                                  const size_t numRows,

                                  const size_t numCols,

                                  const size_t numBins)

{

    const size_t size = numRows*numCols;

    

    max_logLum = reduction(d_logLuminance, size, 1);

    min_logLum = reduction(d_logLuminance, size, 0);

    

    size_t histograma_size = sizeof(unsigned int)*numBins;



    hipMemset(d_cdf, 0, histograma_size);  

    histograma<<<(numCols*numRows + THREADS - 1)/THREADS, THREADS>>>(d_cdf, d_logLuminance, numBins, min_logLum, max_logLum, size);

    scan<<<(numBins + THREADS -1)/THREADS, THREADS>>>(d_cdf, numBins);

    

    

  //TODO

  /*Here are the steps you need to implement

    1) find the minimum and maximum value in the input logLuminance channel

       store in min_logLum and max_logLum

    2) subtract them to find the range

    3) generate a histogram of all the values in the logLuminance channel using

       the formula: bin = (lum[i] - lumMin) / lumRange * numBins

    4) Perform an exclusive scan (prefix sum) on the histogram to get

       the cumulative distribution of luminance values (this should go in the

       incoming d_cdf pointer which already has been allocated for you)       */

}
