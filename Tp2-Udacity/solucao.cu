#include "hip/hip_runtime.h"
#include "utils.h"

#define THREADS 16

__constant__ float cfilter[81];

__global__
void gaussian_blur(uchar4* const input, uchar4* const output,
                   int numRows, int numCols, const int filterWidth)
{
  int x= threadIdx.x + (blockIdx.x*blockDim.x);
  int y = threadIdx.y + (blockIdx.y*blockDim.y);
  int l, m;

  if (x< numCols && y < numRows)
  {
    float s = 0, s2 = 0, s3 = 0;
    x-= filterWidth/2;   y -= filterWidth/2;
    #pragma unroll
    for (int i = 0; i < filterWidth; i++)
    { 
      m = min(max(y +i, 0), numRows-1);
      for (int j = 0; j < filterWidth; j++)
      {
        l = min(max(x+j, 0), numCols-1);
        register int k = i*filterWidth + j;
        register int c = m*numCols + l;
        s+= cfilter[k] * input[c].x;
        s2 += cfilter[k] * input[c].y;
        s3 += cfilter[k] * input[c].z;
      }
    }

    x+= filterWidth/2;   y += filterWidth/2;
    output[x+(y*numCols)].x = s;
    output[x+ (y*numCols)].y = s2;
    output[x+ (y*numCols)].z = s3;
  }
}

//unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                 const float* const h_filter, const size_t filterWidth)
{
  int num = filterWidth * filterWidth;
  //hipMalloc(&d_filter, sizeof(float) * num);
  //hipMemcpy(d_filter, h_filter, sizeof(float) * num, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(cfilter), h_filter, sizeof(float) * num);

}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  
  const dim3 blockSize(THREADS, THREADS);
  const dim3 gridSize((numCols+THREADS-1)/THREADS, (numRows+THREADS-1)/THREADS);
 
  gaussian_blur<<<gridSize, blockSize>>>(d_inputImageRGBA, d_outputImageRGBA, numRows, numCols, filterWidth);
  
}

void cleanup() {
}
